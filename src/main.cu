#include "hip/hip_runtime.h"
// C++ headers
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>

// GLM headers
#include <glm/glm.hpp>

// ImGUI headers
#include "imgui.h"

// My headers (c++)
#include "app.hpp"
#include "Input.hpp"
#include "Shader.hpp"
#include "Surface.hpp"
#include "Window.hpp"
#include "ThrusterPlume.hpp"
#include "Constants.hpp"
#include "Output.hpp"

// My headers (CUDA)
#include "cuda.cuh"
#include "ParticleContainer.cuh"
#include "Triangle.cuh"

using std::vector, std::string;

struct DepositionInfo {
    size_t num_tris = 0;
    vector<string> surf_names;
    vector<size_t> local_indices;
    vector<size_t> global_indices;
    vector<double> areas;
    vector<int> particles_collected;
    vector<double> deposition_rates;
    vector<double> mass_fluxes;
};

int main (int argc, char *argv[]) {
    using namespace constants;
    // Initialize GPU
    device_vector<int> init{0};

    // Read input and open window, if required
    std::string filename = argc > 1 ? argv[1] : "input.toml";
    Input input = read_input(filename);
    auto window = app::initialize(input);

    // construct triangles and set up diagnostics
    host_vector<Triangle> h_triangles;
    host_vector<size_t> h_material_ids;
    host_vector<Material> h_materials;
    DepositionInfo deposition_info{};
    auto &geometry = input.geometry;

    for (size_t id = 0; id < geometry.surfaces.size(); id++) {
        const auto &surf = geometry.surfaces.at(id);
        const auto &mesh = surf.mesh;
        const auto &material = surf.material;

        h_materials.push_back(surf.material);

        auto ind = 0;
        for (const auto &[i1, i2, i3]: mesh.triangles) {
            auto model = surf.transform.get_matrix();
            auto v1 = make_float3(model*glm::vec4(mesh.vertices[i1].pos, 1.0));
            auto v2 = make_float3(model*glm::vec4(mesh.vertices[i2].pos, 1.0));
            auto v3 = make_float3(model*glm::vec4(mesh.vertices[i3].pos, 1.0));

            Triangle tri{v1, v2, v3};
            h_triangles.push_back(tri);
            h_material_ids.push_back(id);
            if (material.collect) {
                auto global_index = static_cast<int>(h_triangles.size()) - 1;
                Triangle tri{v1, v2, v3};
                deposition_info.surf_names.push_back(surf.name);
                deposition_info.areas.push_back(tri.area);
                deposition_info.global_indices.push_back(global_index);
                deposition_info.local_indices.push_back(ind);
                deposition_info.particles_collected.push_back(0);
                deposition_info.deposition_rates.push_back(0);
                deposition_info.mass_fluxes.push_back(0);
                deposition_info.num_tris++;
            }
            ind++;
        }
    }

    if (input.verbosity > 0) std::cout << "Meshes read." << std::endl;

    // Construct BVH on CPU
    host_vector<BVHNode> h_nodes;
    host_vector<size_t> h_triangle_indices;
    Scene h_scene;
    h_scene.build(h_triangles, h_triangle_indices, h_nodes);

    if (input.verbosity > 0) std::cout << "Bounding volume heirarchy constructed." << std::endl;

    // Send mesh data and BVH to GPU.
    device_vector<Triangle> d_triangles = h_triangles;
    device_vector<size_t> d_surface_ids{h_material_ids};
    device_vector<Material> d_materials{h_materials};
    device_vector<int> d_collected(h_triangles.size(), 0);

    Scene d_scene(h_scene);
    device_vector<BVHNode> d_nodes = h_nodes;
    device_vector<size_t> d_triangle_indices = h_triangle_indices;
    d_scene.triangles = thrust::raw_pointer_cast(d_triangles.data());
    d_scene.triangle_indices = thrust::raw_pointer_cast(d_triangle_indices.data());
    d_scene.nodes = thrust::raw_pointer_cast(d_nodes.data());

    if (input.verbosity > 0) std::cout << "Mesh data sent to GPU" << std::endl;

    // Cast initial rays from plume to find where they hit facility geometry
    // Store result in ParticleContainer pc_plume
    host_vector<HitInfo> hits;
    host_vector<float3> hit_positions;
    host_vector<float> num_emit;
    auto plume = input.plume;
    plume.find_hits(input, h_scene, h_materials, h_material_ids, hits, hit_positions, num_emit);

    // Copy plume results to GPU
    device_vector<HitInfo> d_hits{hits};
    device_vector<float> d_num_emit{num_emit};

    // Create particle container for carbon atoms and renderer
    ParticleContainer particles{"carbon", max_particles, 1.0f, 1};
    app::Renderer renderer(input, &h_scene, plume, particles, geometry);

    // Create timing objects
    size_t step = 0;
    app::Timer timer;
    cuda::Event start{}, stop_compute{}, stop_copy{};

    // Create output file for deposition
    Output output("deposition.csv");

    if (input.verbosity > 0) std::cout << "Beginning main loop." << std::endl;

    while ((input.display && window.open) || (!input.display && timer.physical_time < input.max_time_s)) {
        // Draw GUI and set up for this frame
        app::begin_frame(step, input, window, renderer, timer);

        // TODO: can we move this out of main into a different function
        if (input.display) {
            // Table of collected particle amounts
            auto table_flags = ImGuiTableFlags_BordersH;
            ImVec2 bottom_left = ImVec2(0, ImGui::GetIO().DisplaySize.y);
            ImGui::SetNextWindowPos(bottom_left, ImGuiCond_Always, ImVec2(0.0, 1.0));
            ImGui::Begin("Particle collection info", nullptr, app::imgui_flags);
            if (ImGui::BeginTable("Table", 4, table_flags)) {
                ImGui::TableNextRow();
                ImGui::TableNextColumn();
                ImGui::Text("Surface name");
                ImGui::TableNextColumn();
                ImGui::Text("Triangle ID");
                ImGui::TableNextColumn();
                ImGui::Text("Particles collected");
                ImGui::TableNextColumn();
                ImGui::Text("Deposition rate [um/kh]");
                for (int tri = 0; tri < deposition_info.num_tris; tri++) {
                    ImGui::TableNextRow();
                    ImGui::TableNextColumn();
                    ImGui::Text("%s", deposition_info.surf_names[tri].c_str());
                    ImGui::TableNextColumn();
                    ImGui::Text("%i", deposition_info.local_indices[tri]);
                    ImGui::TableNextColumn();
                    ImGui::Text("%d", deposition_info.particles_collected[tri]);
                    ImGui::TableNextColumn();
                    ImGui::Text("%.3f", deposition_info.deposition_rates[tri]);
                }
                ImGui::EndTable();
            }
            ImGui::End();
        }

        // Main computation loop
        if (step > 0 && !app::sim_paused) {
            start.record();

            // Push particles and sputter from surfaces
            particles.evolve(d_scene, d_materials, d_surface_ids, d_collected,
                             d_hits, d_num_emit, input.particle_weight,
                             input.timestep_s);

            // flag particles that are out of bounds
            particles.flag_out_of_bounds(input.chamber_radius_m, input.chamber_length_m);

            // remove particles with negative weight (out of bounds and phantom emitted particles)
            particles.remove_flagged_particles();

            // record stop time
            stop_compute.record();

            // Track particles collected by each triangle flagged 'collect' and compute diagnostics
            for (int id = 0; id < deposition_info.num_tris; id++) {
                // Copy number of particles collected to CPU
                auto d_begin = d_collected.begin() + deposition_info.global_indices[id];
                thrust::copy(d_begin, d_begin + 1, deposition_info.particles_collected.begin() + id);

                // Compute deposition rate and carbon flux
                double mass_carbon = deposition_info.particles_collected[id]*input.particle_weight*carbon.mass*m_u;
                double volume_carbon = mass_carbon/graphite_density;
                double triangle_area = deposition_info.areas[id];
                double layer_thickness_um = volume_carbon/triangle_area*1e6;
                double physical_time_kh = timer.physical_time/3600/1000;
                deposition_info.deposition_rates[id] = layer_thickness_um/physical_time_kh;
                deposition_info.mass_fluxes[id] = mass_carbon / triangle_area / timer.physical_time;
            }

            // Copy particle data back to CPU
            particles.copy_to_cpu();
            stop_copy.record();

            // timing
            double elapsed_compute = cuda::event_elapsed_time(start, stop_compute);
            double elapsed_copy = cuda::event_elapsed_time(start, stop_copy);
            timer.update_averages(elapsed_compute, elapsed_copy);
        }

        // Draw scene
        renderer.draw(input);

        // Finalize frame and increment timestep
        app::end_frame(input, window);
        if (!app::sim_paused) {
            step ++;
            timer.physical_time += input.timestep_s;
        }

        // Write output to console and file at regular intervals, plus one additional when simulation terminates
        if ((!app::sim_paused && timer.should_output()) ||
            (!input.display && timer.physical_time >= input.max_time_s) ||
            (input.display && !window.open)) {

            std::cout << "  Step " << step
                      << ", Simulation time: " << app::print_time(timer.physical_time)
                      << ", Timestep: " << app::print_time(input.timestep_s)
                      << ", Avg. step time: " << timer.dt_smoothed << " ms" << std::endl;

            timer.next_output_time += input.output_interval_s;
        }
    }

    if (input.verbosity > 0) std::cout << "Program terminated successfully." << std::endl;

    return 0;
}
