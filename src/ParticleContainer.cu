#include "hip/hip_runtime.h"
#include <random>

#include <thrust/distance.h>
#include <thrust/partition.h>

#include "ParticleContainer.cuh"
#include "cuda_helpers.cuh"
#include "gl_helpers.hpp"
#include "Constants.hpp"

// Setup RNG
__global__ void k_setup_rng (hiprandState *rng, uint64_t seed) {
  unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(seed, tid, 0, &rng[tid]);
}

ParticleContainer::ParticleContainer (string name, size_t num, double mass, int charge)
  : name(std::move(name)), mass(mass), charge(charge) {

  // Allocate memory on GPU
  d_position.resize(num);
  d_velocity.resize(num);
  d_weight.resize(num);
  d_tmp.resize(num);
  d_rng.resize(num);

  // Set up RNG for later use
  size_t block_size = 512;
  k_setup_rng<<<num/block_size, block_size>>>(thrust::raw_pointer_cast(d_rng.data()), time(nullptr));
  std::cout << "GPU RNG state initialized." << std::endl;
}

void
ParticleContainer::add_particles (const host_vector<float3> &pos, const host_vector<float3> &vel
                                  , const host_vector<float> &w) {
  auto n = static_cast<int>(
    std::min(
      std::min(pos.size(), vel.size()), 
      w.size()
    )
  );
  if (n == 0) return;

  position.resize(num_particles + n);
  velocity.resize(num_particles + n);
  weight.resize(num_particles + n);

  // Copy particles to CPU arrays
  for (size_t i = 0; i < n; i++) {
    auto id = num_particles + i;
    position[id] = pos[i];
    velocity[id] = vel[i];
    weight[id] = w[i];
  }

  // Copy particles to GPU
  thrust::copy(position.begin() + num_particles, position.end(), d_position.begin() + num_particles);
  thrust::copy(velocity.begin() + num_particles, velocity.end(), d_velocity.begin() + num_particles);
  thrust::copy(weight.begin() + num_particles, weight.end(), d_weight.begin() + num_particles);

  num_particles += n;
}

void ParticleContainer::copy_to_cpu () {
  position = host_vector<float3>(d_position.begin(), d_position.begin() + num_particles);
  velocity = host_vector<float3>(d_velocity.begin(), d_velocity.begin() + num_particles);
  weight = host_vector<float>(d_weight.begin(), d_weight.begin() + num_particles);
}

void ParticleContainer::set_buffers () {
  // enable buffer
  this->mesh.set_buffers();
  glGenBuffers(1, &this->buffer);
}

void ParticleContainer::draw () {

  // Bind vertex array
  auto vao = this->mesh.vao;
  GL_CHECK(glBindVertexArray(vao));

  // Send over model matrix data
  auto mat_vector_size = static_cast<GLsizei>(this->num_particles*sizeof(vec3));
  GL_CHECK(glBindBuffer(GL_ARRAY_BUFFER, this->buffer));
  GL_CHECK(glBufferData(GL_ARRAY_BUFFER, mat_vector_size, &position[0], GL_DYNAMIC_DRAW));

  // Set attribute pointers for translation
  GL_CHECK(glEnableVertexAttribArray(2));
  GL_CHECK(glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, sizeof(vec3), nullptr));
  GL_CHECK(glVertexAttribDivisor(2, 1));

  // Bind element array buffer
  GL_CHECK(glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->mesh.ebo));

  // Draw meshes
  GL_CHECK(glDrawElementsInstanced(GL_TRIANGLES, static_cast<unsigned int>(3*this->mesh.num_triangles), GL_UNSIGNED_INT
                                   , nullptr, num_particles));

  // unbind buffers
  GL_CHECK(glBindVertexArray(0));
  GL_CHECK(glBindBuffer(GL_ARRAY_BUFFER, 0));
  GL_CHECK(glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0));
}


__host__ __device__ float carbon_diffuse_prob (float cos_incident_angle, float incident_energy_ev) {
  // fit parameters
  constexpr auto angle_offset = 1.6823f;
  constexpr auto energy_offset = 65.6925f;
  constexpr auto energy_scale = 34.5302f;

  auto fac = (cos_incident_angle - angle_offset)*logf((incident_energy_ev + energy_offset)/energy_scale);
  auto diffuse_coeff = 0.003f + fac*fac;
  return diffuse_coeff;
}

__device__ float3
sample_diffuse (const Triangle &tri, const float3 norm, float thermal_speed, hiprandState *rng) {
  // sample from a cosine distribution
//  auto c_tan1 = hiprand_normal(rng);
//  auto c_tan2 = hiprand_normal(rng);
//  auto c_norm = abs(hiprand_normal(rng));
  using namespace constants;

  auto samples = hiprand_normal2(rng);
  auto unif = hiprand_uniform(rng);

  auto c_norm = sqrt(-0.5*log(unif));
  auto c_tan1 = samples.x;
  auto c_tan2 = samples.y;

  // get tangent vectors
  // TODO: may be worth pre-computing these?
  auto tan1 = normalize(tri.v1 - tri.v0);
  auto tan2 = cross(tan1, norm);

  // Compute new velocity vector
  auto vel_refl = thermal_speed*(c_norm*norm + c_tan1*tan1 + c_tan2*tan2);
  return vel_refl;
}

DeviceParticleContainer ParticleContainer::data () {
  DeviceParticleContainer pc;
  pc.position = thrust::raw_pointer_cast(this->d_position.data());
  pc.velocity = thrust::raw_pointer_cast(this->d_velocity.data());
  pc.weight = thrust::raw_pointer_cast(this->d_weight.data());
  pc.rng = thrust::raw_pointer_cast(this->d_rng.data());
  pc.num_particles = this->num_particles;
  return pc;
}

__global__ void
k_evolve (DeviceParticleContainer pc
          , const Triangle *tris, const size_t num_triangles
          , const Material *materials, const size_t *material_ids
          , int *collected
          , const HitInfo *hits, const float *emit_prob, size_t num_hits
          , float input_weight, float dt) {

  // Thread ID, i.e. what particle we're currently moving
  unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;

  using namespace constants;

  // Particle mass
  // FIXME: currently hard-coded to carbon, easy to fix by passing in mass as a param
  const double mass = 12.011*m_u;

  // Particle energy
  const double energy_factor = 0.5*mass/q_e;

  // k_B / m_u (for thermal speed calculations)
  const auto thermal_speed_factor = static_cast<float>(sqrt(k_b/mass));

  // Push particles
  if (tid < pc.num_particles) {

    auto pos = pc.position[tid];
    auto vel = pc.velocity[tid];

    // Check for intersections with boundaries
    Ray ray{.origin = pos, .direction = dt*vel};
    auto closest_hit = ray.cast(tris, num_triangles);

    if (closest_hit.t <= 1) {
      auto &[_, t, hit_pos, norm, hit_triangle_id] = closest_hit;

      // Get material info where we hit
      auto &mat = materials[material_ids[hit_triangle_id]];

      // Generate a random number
      auto local_rng = &pc.rng[tid];
      auto uniform = hiprand_uniform(local_rng);

      // get incident angle and energy
      auto velnorm_2 = dot(vel, vel);
      auto cos_incident_angle = abs(dot(vel, -norm)/sqrt(velnorm_2));
      auto incident_energy_ev = static_cast<float>(energy_factor*velnorm_2);

      // Get sticking and diffuse coeff from model
      auto diffuse_coeff = carbon_diffuse_prob(cos_incident_angle, incident_energy_ev);
      auto sticking_coeff = 1.0f - diffuse_coeff;

      if (uniform < sticking_coeff) {
        // Particle sticks to surface
        pc.position[tid] = hit_pos;
        pc.velocity[tid] = float3{0.0f, 0.0f, 0.0f};

        // Record that we hit this triangle
        atomicAdd(&collected[hit_triangle_id], 1);

        // set weight negative to flag for removal
        // magnitude indicates which triangle we hit
        pc.weight[tid] = static_cast<float>(-hit_triangle_id);

      } else if (uniform < diffuse_coeff + sticking_coeff) {
        // Particle reflects diffusely based on surface temperature
        // TODO: pass thermal speed (or sqrt of temperature) instead of temperature to avoid this
        //
        auto sqrt_temp = sqrtf(mat.temperature_k);
        auto thermal_speed = thermal_speed_factor*sqrt_temp;
        auto vel_refl = sample_diffuse(tris[hit_triangle_id], norm, thermal_speed, local_rng);

        // Get particle position
        // (assuming particle reflects ~instantaneously then travels according to new velocity vector)
        // TODO: most of this code is shared with below--worth unifying?
        auto final_pos = hit_pos + (1 - t)*dt*vel_refl;
        pc.position[tid] = final_pos;
        pc.velocity[tid] = vel_refl;
      } else {
        // Particle reflects specularly
        float3 vel_norm = dot(vel, norm)*norm;
        float3 vel_refl = vel - 2*vel_norm;

        auto final_pos = hit_pos + (1 - t)*dt*vel_refl;
        pc.position[tid] = final_pos;
        pc.velocity[tid] = vel_refl;
      }
    } else {
      pc.position[tid] = pos + dt*vel;
    }
  } else if (tid < num_hits + pc.num_particles) {
    // Emit new particles
    auto &hit = hits[tid - pc.num_particles];
    auto p_emit = emit_prob[tid - pc.num_particles];
    auto local_rng = &pc.rng[tid];

    // compute diffuse velocity
    auto &tri = tris[hit.id];
    auto &mat = materials[material_ids[hit.id]];
    auto thermal_speed = sqrtf(mat.temperature_k)*thermal_speed_factor;
    auto vel = sample_diffuse(tri, hit.norm, thermal_speed, local_rng);

    // generate rng
    auto u = hiprand_uniform(local_rng);

    // add new particles (negative weight if not real)
    if (u < p_emit*dt) {
      pc.position[tid] = hit.pos + 1e-2*dt*vel;
      pc.velocity[tid] = vel;
      pc.weight[tid] = input_weight;
    } else {
      pc.position[tid] = -1000.000*hit.pos;
      pc.velocity[tid] = {0.0, 0.0, 0.0};
      pc.weight[tid] = -1.0;
    }
  }
}

std::pair<dim3, dim3> ParticleContainer::get_kernel_launch_params (size_t num_elems, size_t block_size) const {
  auto grid_size = static_cast<int>(ceil(static_cast<float>(num_elems)/static_cast<float>(block_size)));
  dim3 grid(grid_size, 1, 1);
  dim3 block(block_size, 1, 1);
  return std::make_pair(grid, block);
}

void ParticleContainer::evolve (const device_vector<Triangle> &tris
                                , const device_vector<Material> &mats, const device_vector<size_t> &ids
                                , device_vector<int> &collected
                                , const device_vector<HitInfo> &hits, const device_vector<float> &num_emit
                                , const float input_weight, const float dt) {


  // TODO: could move all of the device geometric info into a struct
  auto d_tri_ptr = thrust::raw_pointer_cast(tris.data());
  auto d_id_ptr = thrust::raw_pointer_cast(ids.data());
  auto d_mat_ptr = thrust::raw_pointer_cast(mats.data());

  auto d_col_ptr = thrust::raw_pointer_cast(collected.data());
  auto d_hit_ptr = thrust::raw_pointer_cast(hits.data());
  auto d_emit_ptr = thrust::raw_pointer_cast(num_emit.data());

  auto [grid, block] = get_kernel_launch_params(num_particles + hits.size());

  k_evolve<<<grid, block>>>(
    this->data()
    , d_tri_ptr, tris.size()
    , d_mat_ptr, d_id_ptr, d_col_ptr
    , d_hit_ptr, d_emit_ptr, hits.size()
    , input_weight, dt);

  this->num_particles += hits.size();

  hipDeviceSynchronize();
}

float rand_uniform (float min, float max) {
  static std::default_random_engine rng;

  std::uniform_real_distribution<float> dist(min, max);
  return dist(rng);
}

float rand_normal (float mean, float std) {
  static std::default_random_engine rng;

  std::normal_distribution<float> dist(mean, std);
  return dist(rng);
}

void ParticleContainer::emit (Triangle &triangle, Emitter emitter, float dt) {
  auto num_emit = emitter.flux*triangle.area*dt;
  int num_emit_int = static_cast<int>(num_emit);
  auto remainder = num_emit - static_cast<float>(num_emit_int);

  auto u = rand_uniform();
  if (u < remainder) {
    num_emit_int += 1;
  }

  if (num_emit_int < 1) {
    return;
  }

  host_vector<float3> pos(num_emit_int);
  host_vector<float3> vel(num_emit_int);
  host_vector<float> w(num_emit_int, 1.0f);

  for (int i = 0; i < num_emit_int; i++) {
    auto pt = triangle.sample(rand_uniform(), rand_uniform());
    auto norm = emitter.reverse ? -triangle.norm : triangle.norm;
    // offset particle very slightly by norm
    auto tol = 0.0001f;
    pos[i] = pt + tol*norm;
    auto jitter = float3{
      rand_normal(0, emitter.spread), rand_normal(0, emitter.spread), rand_normal(0, emitter.spread)};
    vel[i] = emitter.velocity*(norm + jitter);
  }

  add_particles(pos, vel, w);
}

__global__ void k_flag_oob (float3 *pos, float *weight, float radius2, float halflength, size_t n) {
  unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;
  if (id < n && weight[id] > 0) {
    auto r = pos[id];
    auto dist2 = r.x*r.x + r.y*r.y;
    if (dist2 > radius2 || r.z < -halflength || r.z > halflength) {
      // Particles that are oob get negative weight
      weight[id] = -1;
    }
  }
}

void ParticleContainer::flag_out_of_bounds (float radius, float length) {
  auto [grid, block] = get_kernel_launch_params(num_particles);

  auto d_pos_ptr = thrust::raw_pointer_cast(d_position.data());
  auto d_wgt_ptr = thrust::raw_pointer_cast(d_weight.data());
  k_flag_oob<<<grid, block>>>(d_pos_ptr, d_wgt_ptr, radius*radius, length/2, num_particles);
  hipDeviceSynchronize();
}

struct IsPositive {
  __host__ __device__ bool operator() (const float &w) {
    return w > 0;
  }
};

void ParticleContainer::remove_flagged_particles () {
  // reorder positions and velocities so that particles with negative weight follow those with positive weight
  thrust::partition(d_position.begin(), d_position.begin() + num_particles, d_weight.begin(), IsPositive());
  thrust::partition(d_velocity.begin(), d_velocity.begin() + num_particles, d_weight.begin(), IsPositive());

  // reorder weights according to the same scheme as above
  // copy weights to temporary vector first
  // thrust partition likely is allocating some temporary memory
  // to avoid this, we would probably want to set up a custom allocator
  // c.f. https://github.com/NVIDIA/thrust/blob/1.6.0/examples/cuda/custom_temporary_allocation.cu
  // Alternatively, could use CUB device partition, which gives us more control to allocate temporary data
  // c.f. https://nvidia.github.io/cccl/cub/api/structcub_1_1DevicePartition.html#_CPPv4N3cub15DevicePartitionE
  thrust::copy(d_weight.begin(), d_weight.begin() + num_particles, d_tmp.begin());
  auto ret = thrust::partition(d_weight.begin(), d_weight.begin() + num_particles, d_tmp.begin(), IsPositive());

  // Reset number of particles to the middle of the partition
  num_particles = static_cast<int>(thrust::distance(d_weight.begin(), ret));
}


std::ostream &operator<< (std::ostream &os, ParticleContainer const &pc) {
  os << "==========================================================\n";
  os << "Particle container \"" << pc.name << "\"\n";
  os << "==========================================================\n";
  os << "Mass: " << pc.mass << "\n";
  os << "Charge: " << pc.charge << "\n";
  os << "Number of particles: " << pc.num_particles << "\n";
  os << "----------------------------------------------------------\n";
  os << "\tx\ty\tz\tvx\tvy\tvz\tw\t\n";
  os << "----------------------------------------------------------\n";
  for (int i = 0; i < pc.num_particles; i++) {
    os << "\t" << pc.position[i].x << " ";
    os << pc.position[i].y << "  ";
    os << pc.position[i].z << "  ";
    os << pc.velocity[i].x << "  ";
    os << pc.velocity[i].y << "  ";
    os << pc.velocity[i].z << "  ";
    os << pc.weight[i] << "\n";
  }
  os << "==========================================================\n";

  return os;
}