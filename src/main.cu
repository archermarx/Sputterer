#include "hip/hip_runtime.h"
// C++ headers
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>

// GLM headers
#include <glm/glm.hpp>

// ImGUI headers
#include "imgui.h"

// My headers (c++)
#include "app.hpp"
#include "Input.hpp"
#include "Shader.hpp"
#include "Surface.hpp"
#include "Window.hpp"
#include "ThrusterPlume.hpp"
#include "Constants.hpp"

// My headers (CUDA)
#include "cuda.cuh"
#include "ParticleContainer.cuh"
#include "Triangle.cuh"

using std::vector, std::string;

string print_time (double time_s) {
  char buf[64];
  int factor = 1;
  string str = "s";

  if (time_s < 1e-6) {
    factor = 1'000'000'000;
    str = "ns";
  } else if (time_s < 1e-3) {
    factor = 1'000'000;
    str = "us";
  } else if (time_s < 1) {
    factor = 1000;
    str = "ms";
  }

  sprintf(buf, "%.3f %s", time_s*factor, str.c_str());

  return {buf};
}

int main (int argc, char *argv[]) {
  // Handle command line arguments
  string filename{"../input.toml"};
  bool display{true};

  if (argc > 1) {
    filename = argv[1];
  }
  if (argc > 2) {
    display = static_cast<bool>(std::stoi(argv[2]));
  }

  using namespace constants;

  Input input(filename);
  input.read();

  std::cout << "Input read." << std::endl;

  app::camera.orientation = glm::normalize(glm::vec3(input.chamber_radius));
  app::camera.distance = 2.0f*input.chamber_radius;
  app::camera.yaw = -135;
  app::camera.pitch = 30;
  app::camera.update_vectors();

  // Create particle container
  ParticleContainer pc{"noname", max_particles, 1.0f, 1};

  // construct triangles
  host_vector <Triangle> h_triangles;

  host_vector <size_t> h_material_ids;
  host_vector <Material> h_materials;

  host_vector<char> h_to_collect;
  std::vector<int> collect_inds_global;
  std::vector<int> collect_inds_local;

  std::vector<string> surface_names;

  for (size_t id = 0; id < input.surfaces.size(); id++) {
    const auto &surf = input.surfaces.at(id);
    const auto &mesh = surf.mesh;
    const auto &material = surf.material;

    surface_names.push_back(surf.name);
    h_materials.push_back(surf.material);

    auto ind = 0;
    for (const auto &[i1, i2, i3]: mesh.triangles) {
      auto model = surf.transform.get_matrix();
      auto v1 = make_float3(model*glm::vec4(mesh.vertices[i1].pos, 1.0));
      auto v2 = make_float3(model*glm::vec4(mesh.vertices[i2].pos, 1.0));
      auto v3 = make_float3(model*glm::vec4(mesh.vertices[i3].pos, 1.0));

      h_triangles.push_back({v1, v2, v3});
      h_material_ids.push_back(id);
      if (material.collect) {
        collect_inds_global.push_back(static_cast<int>(h_triangles.size()) - 1);
        collect_inds_local.push_back(ind);
      }
      ind++;
    }
  }

  host_vector<int> collected(collect_inds_global.size(), 0);
  std::cout << "Meshes read." << std::endl;

  // Send mesh data to GPU. Really slow for some reason (multiple seconds)!
  device_vector <Triangle> d_triangles = h_triangles;
  device_vector <size_t> d_surface_ids{h_material_ids};
  device_vector <Material> d_materials{h_materials};
  device_vector<int> d_collected(h_triangles.size(), 0);

  std::cout << "Mesh data sent to GPU" << std::endl;

  // Create plume model
  ThrusterPlume plume{};
  plume.location = input.plume_origin;
  plume.direction = glm::normalize(input.plume_direction);
  plume.beam_current = input.ion_current_a;
  plume.background_pressure = input.background_pressure_torr;
  plume.model_params = input.plume_model_params;
  plume.beam_energy_ev = input.beam_energy_ev;
  plume.scattered_energy_ev = input.scattered_energy_ev;
  plume.cex_energy_ev = input.cex_energy_ev;

  // Display objects
  Window window{.name = "Sputterer", .width = app::screen_width, .height = app::screen_height};
  Shader mesh_shader{}, particle_shader{}, plume_shader{};
  if (display) {
    // enable window
    window.enable();

    // Register window callbacks
    glfwSetCursorPosCallback(window.window, app::mouse_cursor_callback);
    glfwSetScrollCallback(window.window, app::scroll_callback);

    window.initialize_imgui();

    // Load mesh shader
    mesh_shader.load("../shaders/shader.vert", "../shaders/shader.frag");

    // initialize mesh buffers
    for (auto &surf: input.surfaces) {
      surf.mesh.set_buffers();
    }

    // Load particle shader
    particle_shader.load("../shaders/particle.vert", "../shaders/particle.frag");
    particle_shader.use();
    constexpr vec3 particle_scale{0.01f};
    particle_shader.set_vec3("scale", particle_scale);

    // Set up particle mesh
    pc.mesh.read_from_obj("../o_sphere.obj");
    pc.set_buffers();

    // Load plume shader
    plume_shader.load("../shaders/plume.vert", "../shaders/plume.frag", "../shaders/plume.geom");
    plume_shader.use();
    float plume_length = input.chamber_length/2 - plume.location.z;
    plume_shader.set_float("length", plume_length);
    plume_shader.set_vec3("direction", plume.direction);
    plume.set_buffers();
  }

  // Create timing objects
  size_t frame = 0;

  float avg_time_compute = 0.0f, avg_time_total = 0.0f;
  float iter_reset = 25;
  float time_const = 1/iter_reset;
  double physical_time = 0;
  float delta_time_smoothed = 0;

  auto next_output_time = 0.0f;

  cuda::Event start{}, stop_compute{}, stop_copy{};

  auto current_time = std::chrono::system_clock::now();
  auto last_time = std::chrono::system_clock::now();

  // Create output file for deposition
  string output_filename{"deposition.csv"};
  std::ofstream output_file;
  output_file.open(output_filename);
  output_file << "Time(s),Surface name,Local triangle ID,Global triangle ID,Macroparticles collected,Mass collected"
              << std::endl;
  output_file.close();

  // Cast initial rays from plume
  int num_rays = 50'000;
  host_vector <HitInfo> hits;
  host_vector <float3> hit_positions;
  vector<float> num_emit;
  host_vector <float3> vel;
  host_vector<float> ws;

  float max_emit = 0.0;

  // plume coordinate system
  auto up = vec3{0.0, 1.0, 0.0};
  auto right = cross(plume.direction, up);
  up = cross(right, plume.direction);

  auto incident = constants::xenon;
  auto target = constants::carbon;

  auto [main_fraction, scattered_fraction, _] = plume.current_fractions();
  auto beam_fraction = main_fraction + scattered_fraction;
  main_fraction = main_fraction/beam_fraction;


  for (int i = 0; i < num_rays; i++) {

    // select whether ray comes from main beam or scattered beam based on
    // fraction of beam that is scattered vs main
    auto u = rand_uniform();
    double div_angle{};
    if (u < main_fraction) {
      div_angle = plume.main_divergence_angle();
    } else {
      div_angle = plume.scattered_divergence_angle();
    }

    auto azimuth = rand_uniform(0, 2*constants::pi);
    auto elevation = abs(rand_normal(0, div_angle/sqrt(2.0)));

    auto direction = cos(elevation)*plume.direction + sin(elevation)*(cos(azimuth)*right + sin(azimuth)*up);
    Ray r{.origin = make_float3(plume.location + direction*1e-3f), .direction = normalize(make_float3(direction))};
    auto hit = r.cast(h_triangles.data(), h_triangles.size());
    if (hit.hits) {
      auto hit_pos = r.at(hit.t);
      hits.push_back(hit);
      hit_positions.push_back(hit_pos);
      vel.push_back({0.0f, 0.0f, 0.0f});
      ws.push_back(0.0f);

      auto cos_hit_angle = static_cast<double>(dot(r.direction, -hit.norm));
      auto hit_angle = acos(cos_hit_angle);

      auto yield = sputtering_yield(plume.beam_energy_ev, hit_angle, incident, target);
      auto n_emit = yield*plume.beam_current*beam_fraction/constants::q_e/num_rays/input.particle_weight;
      if (n_emit > max_emit)
        max_emit = n_emit*input.timestep_s;
      num_emit.push_back(n_emit);
    }
  }

  if (max_emit > 1.0) {
    std::cout << "WARNING: decreasing timestep so that max 1 particle emitted per location per timestep" << std::endl;
    input.timestep_s /= max_emit;
  }
  std::cout << "Max emission probability: " << max_emit << std::endl;

  ParticleContainer pc_plume{"plume", hit_positions.size()};
  pc_plume.add_particles(hit_positions, vel, ws);
  if (display) {
    pc_plume.mesh.read_from_obj("../o_sphere.obj");
    pc_plume.set_buffers();
  }

  device_vector <HitInfo> d_hits{hits};
  device_vector<float> d_num_emit{num_emit};

  std::cout << "Beginning main loop." << std::endl;

  bool render_plume_cone = true;
  bool render_plume_particles = true;
  bool plume_on = false;

  while ((display && window.open) || (!display && physical_time < input.max_time_s)) {

    if (display) {
      Window::begin_render_loop();

      // Timing info
      auto flags = ImGuiWindowFlags_NoMove | ImGuiWindowFlags_NoScrollbar | ImGuiWindowFlags_NoResize |
                   ImGuiWindowFlags_NoInputs | ImGuiWindowFlags_NoTitleBar | ImGuiWindowFlags_NoSavedSettings;
      float padding = 0.0f;
      ImVec2 bottom_right = ImVec2(ImGui::GetIO().DisplaySize.x - padding, ImGui::GetIO().DisplaySize.y - padding);
      ImGui::SetNextWindowPos(bottom_right, ImGuiCond_Always, ImVec2(1.0, 1.0));
      ImGui::Begin("Frame time", nullptr, flags);
      ImGui::Text("Simulation step %li (%s)\nSimulation time: %s\nCompute time: %.3f ms (%.2f%% data "
                  "transfer)   \nFrame time: %.3f ms (%.1f fps, %.2f%% compute)   \nParticles: %i", frame, print_time(
          input.timestep_s).c_str(), print_time(physical_time).c_str(), avg_time_compute,
        (1.0f - avg_time_compute/avg_time_total)*100, delta_time_smoothed, 1000/delta_time_smoothed,
        (avg_time_total/delta_time_smoothed)*100, pc.num_particles);
      ImGui::End();

      // Table of collected particle amounts
      auto table_flags = ImGuiTableFlags_BordersH;
      ImVec2 bottom_left = ImVec2(0, ImGui::GetIO().DisplaySize.y - padding);
      ImGui::SetNextWindowPos(bottom_left, ImGuiCond_Always, ImVec2(0.0, 1.0));
      ImGui::Begin("Particle collection info", nullptr, flags);
      if (ImGui::BeginTable("Table", 4, table_flags)) {
        ImGui::TableNextRow();
        ImGui::TableNextColumn();
        ImGui::Text("Surface name");
        ImGui::TableNextColumn();
        ImGui::Text("Triangle ID");
        ImGui::TableNextColumn();
        ImGui::Text("Particles collected");
        ImGui::TableNextColumn();
        ImGui::Text("Mass collected [kg]");
        for (int row = 0; row < collect_inds_global.size(); row++) {
          auto triangle_id = collect_inds_global[row];
          ImGui::TableNextRow();
          ImGui::TableNextColumn();
          ImGui::Text("%s", surface_names.at(h_material_ids[triangle_id]).c_str());
          ImGui::TableNextColumn();
          ImGui::Text("%i", static_cast<int>(collect_inds_local[row]));
          ImGui::TableNextColumn();
          ImGui::Text("%d", collected[row]);
          ImGui::TableNextColumn();
          ImGui::Text("%.3e", static_cast<double>(collected[row]*carbon.mass*m_u*input.particle_weight));
        }
        ImGui::EndTable();
      }
      ImGui::End();

      flags = ImGuiWindowFlags_NoMove | ImGuiWindowFlags_NoScrollbar | ImGuiWindowFlags_NoResize
              | ImGuiWindowFlags_NoTitleBar | ImGuiWindowFlags_NoSavedSettings;

      ImVec2 top_right = ImVec2(ImGui::GetIO().DisplaySize.x - padding, 0);
      ImGui::SetNextWindowPos(top_right, ImGuiCond_Always, ImVec2(1.0, 0.0));
      ImGui::Begin("Options", nullptr, flags);
      if (ImGui::BeginTable("split", 2)) {
        ImGui::TableNextColumn();
        ImGui::Checkbox("Show plume particles", &render_plume_particles);
        ImGui::TableNextColumn();
        ImGui::Checkbox("Show plume cone", &render_plume_cone);
      }
      ImGui::EndTable();
      ImGui::End();
    }

    // Record iteration timing information
    current_time = std::chrono::system_clock::now();
    app::delta_time =
      static_cast<float>(std::chrono::duration_cast<std::chrono::microseconds>(current_time - last_time).count())/
      1e6;
    last_time = current_time;

    // set physical timestep_s. if we're displaying a window, we set the physical timestep_s based on the rendering
    // timestep_s to get smooth performance at different window sizes. If not, we just use the user-provided timestep_s
    if (plume_on) {
      physical_time += input.timestep_s;
    }
    delta_time_smoothed = (1 - time_const)*delta_time_smoothed + time_const*app::delta_time*1000;

    // Main computations
    if (frame > 0 && plume_on) {
      start.record();

      // Emit particles
      size_t tri_count{0};
      for (const auto &surf: input.surfaces) {
        auto &emitter = surf.emitter;
        if (!emitter.emit) {
          continue;
        }

        for (size_t i = 0; i < surf.mesh.num_triangles; i++) {
          pc.emit(h_triangles[i], emitter, input.timestep_s);
        }
        tri_count += surf.mesh.num_triangles;
      }

      // Push particles and sputter from surfaces
      pc.evolve(d_triangles, d_materials, d_surface_ids, d_collected, d_hits, d_num_emit, input.particle_weight
                , input.timestep_s);

      // Remove particles that are out of bounds
      pc.flag_out_of_bounds(input.chamber_radius, input.chamber_length);

      // remove particles with negative weight (out of bounds and phantom emitted particles)
      pc.remove_flagged_particles();

      // record stop time
      stop_compute.record();

      // Track particles collected by each triangle flagged 'collect'
      for (int id = 0; id < collect_inds_global.size(); id++) {
        auto d_begin = d_collected.begin() + collect_inds_global[id];
        thrust::copy(d_begin, d_begin + 1, collected.begin() + id);
      }

      // Copy particle data back to CPU
      pc.copy_to_cpu();

      stop_copy.record();

      float elapsed_compute, elapsed_copy;
      elapsed_compute = cuda::event_elapsed_time(start, stop_compute);
      elapsed_copy = cuda::event_elapsed_time(start, stop_copy);

      avg_time_compute = (1 - time_const)*avg_time_compute + time_const*elapsed_compute;
      avg_time_total = (1 - time_const)*avg_time_total + time_const*elapsed_copy;
    }

    // Rendering
    if (display) {

      // get camera matrix for use in particle and plume shaders
      auto cam = app::camera.get_projection_matrix(app::aspect_ratio)*app::camera.get_view_matrix();

      // 1. draw user-provided geometry

      // update update camera uniforms
      mesh_shader.use();
      mesh_shader.update_view(app::camera, app::aspect_ratio);

      for (const auto &surface: input.surfaces) {
        // set the model matrix and object color per surface
        mesh_shader.use();
        mesh_shader.set_mat4("model", surface.transform.get_matrix());
        mesh_shader.set_vec3("objectColor", surface.color);
        surface.mesh.draw();
      }

      // 2. draw particles (instanced!)
      if (plume_on && pc.num_particles > 0) {
        // activate particle shader
        particle_shader.use();

        constexpr vec3 particle_color{0.05f};
        // send camera information to shader
        particle_shader.set_vec3("objectColor", particle_color);
        particle_shader.set_mat4("camera", cam);

        // draw particles
        pc.draw();
      }

      // Draw plume particles
      if (render_plume_particles) {
        particle_shader.use();
        particle_shader.set_vec3("objectColor", {0.2, 0.75, 0.94});
        particle_shader.set_mat4("camera", cam);
        pc_plume.draw();
      }

      if (render_plume_cone) {
        // 3. draw plume cones
        plume_shader.use();
        plume_shader.set_mat4("camera", cam);

        // draw main beam
        auto div_angle = plume.main_divergence_angle();
        plume_shader.set_bool("main_beam", true);
        plume_shader.set_float("angle", div_angle);
        plume.draw();

        div_angle = plume.scattered_divergence_angle();
        plume_shader.set_bool("main_beam", false);
        plume_shader.set_float("angle", div_angle);
        plume.draw();
      }
    }

    if (plume_on && physical_time > next_output_time || (!display && physical_time >= input.max_time_s) ||
        (display && !window.open)) {
      // Write output to console at regular intervals, plus one additional when simulation terminates
      std::cout << "Step " << frame << ", Simulation time: " << print_time(physical_time)
                << ", Timestep: " << print_time(input.timestep_s) << ", Avg. step time: " << delta_time_smoothed
                << " ms" << std::endl;

      // Log deposition rate info
      output_file.open(output_filename, std::ios_base::app);
      for (int i = 0; i < collect_inds_global.size(); i++) {
        auto triangle_id_global = collect_inds_global[i];
        output_file << physical_time << ",";
        output_file << surface_names.at(h_material_ids[triangle_id_global]) << ",";
        output_file << collect_inds_local.at(i) << ",";
        output_file << triangle_id_global << ",";
        output_file << collected[i] << ",";
        output_file << collected[i]*input.particle_weight*constants::carbon.mass*constants::m_u << "\n";
      }
      output_file.close();

      next_output_time += input.output_interval;
    }

    if (display) {
      window.end_render_loop();
      app::process_input(window.window);
    }

    frame += 1;
  }

  std::cout << "Program terminated successfully." << std::endl;

  return 0;
}
